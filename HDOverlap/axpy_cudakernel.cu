//******************************************************************************************************************//
// Copyright (c) 2021, University of North Carolina at Charlotte
// and Lawrence Livermore National Security, LLC.
// SPDX-License-Identifier: (BSD-3-Clause)
//*****************************************************************************************************************//
// Experimental test for new function memcpy_async in CUDA11
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>
#include <cmath>
#include <omp.h>
#include <vector>
#include "hip/hip_runtime.h"


double read_timer_ms() {
    struct timeb tm;
    ftime(&tm);
    return (double) tm.time * 1000.0 + (double) tm.millitm;
}

/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 //use a fixed number for now
/* zero out the entire vector */
void zero(REAL *A, int n)
{
    int i;
    for (i = 0; i < n; i++) {
        A[i] = 0.0;
    }
}

__global__ 
void
axpy_cudakernel_1perThread(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > 0 &&i < n) y[i] += a*x[i];
}

double axpy_cuda_normal(REAL* x, REAL* y, int n, REAL a) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));
  double time = read_timer_ms();

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice);
  time = read_timer_ms() - time;

  // Perform axpy elements
  axpy_cudakernel_1perThread<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();
  
  hipMemcpy(y, d_y, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
  return time;
}

double axpy_cuda_async(REAL* x, REAL* y, int n, REAL a) {
		hipStream_t stream1;
		hipError_t result;
		result = hipStreamCreate(&stream1);

  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));
  double time2 = read_timer_ms();

  hipMemcpyAsync(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice, stream1);
  time2 = read_timer_ms() - time2;


  //cudaMemcpy(d_x, x, n*sizeof(REAL), cudaMemcpyHostToDevice);
  //cudaMemcpy(d_y, y, n*sizeof(REAL), cudaMemcpyHostToDevice);
    // Perform axpy elements
  axpy_cudakernel_1perThread<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();
  

  hipMemcpy(y, d_y, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
return time2;
}



/* initialize a vector with random floating point numbers */
void init(REAL *A, int n)
{
    int i;
    for (i = 0; i < n; i++) {
        A[i] = (double)drand48();
    }
}

/*serial version */
void axpy(REAL* x, REAL* y, long n, REAL a) {
  int i;
  for (i = 1; i < n; ++i)
  {
    y[i] += a * x[i];
  }
}

/* compare two arrays and return percentage of difference */
REAL check(REAL*A, REAL*B, int n)
{
    int i;
    REAL diffsum =0.0, sum = 0.0;
    for (i = 0; i < n; i++) {
        diffsum += fabs(A[i] - B[i]);
        sum += fabs(B[i]);
    }
    return diffsum/sum;
}

int main(int argc, char *argv[])
{
  int n;
  REAL *y_cuda, *y, *x, *y_cuda_async;
  REAL a = 123.456;

  n = VEC_LEN;
  fprintf(stderr, "Usage: axpy <n>\n");
  if (argc >= 2) {
    n = atoi(argv[1]);
  }
  y_cuda = (REAL *) malloc(n * sizeof(REAL));
  y_cuda_async = (REAL *) malloc(n * sizeof(REAL));
  y  = (REAL *) malloc(n * sizeof(REAL));
  x = (REAL *) malloc(n * sizeof(REAL));

  srand48(1<<12);
  init(x, n);
  init(y_cuda, n);
  memcpy(y, y_cuda, n*sizeof(REAL));
  memcpy(y_cuda_async, y_cuda, n*sizeof(REAL));

  int i;
  int num_runs = 10;
  for (i=0; i<num_runs; i++) axpy(x, y, n, a);

  //warming up
  axpy_cuda_normal(x, y_cuda_async, n, a);
  axpy_cuda_async(x, y_cuda_async, n, a);

  /* cuda version */
  double elapsed;// = read_timer_ms();
  for (i=0; i<num_runs; i++) elapsed += axpy_cuda_normal(x, y_cuda_async, n, a);
  elapsed =  elapsed/num_runs;

  double elapsed1;// = read_timer_ms();
  for (i=0; i<num_runs; i++) elapsed1 += axpy_cuda_async(x, y_cuda_async, n, a);
  elapsed1 = elapsed1/num_runs;

  REAL checkresult = check(y_cuda, y, n);
  REAL checkresult1 = check(y_cuda_async, y, n);

  printf("axpy(%d): checksum: %g, time: %0.2fms\n", n, checkresult, elapsed);
  printf("axpy_async(%d): checksum: %g, time: %0.2fms\n", n, checkresult1, elapsed1);

  //assert (checkresult < 1.0e-10);

  free(y_cuda);
  free(y);
  free(x);
  return 0;
}
