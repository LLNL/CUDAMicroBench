#include "hip/hip_runtime.h"
#include "SpMM_csr.h"

__global__ void spmm_csr_kernel(const int num_rows, const int *ptrA, const int * indicesA, const REAL *dataA, const int *ptrB, const int * indicesB, const REAL *dataB,  REAL* result, int nnzA, int nnzB)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < num_rows){

        int row_start = ptrA[row];
        int row_end = ptrA[row+1];

        for(int k =0; k<num_rows; k++){ //iterate over B column
          float dot = 0;
          for ( int i = row_start; i < row_end; i++) {
            //int colNum = k;  //The col of the element
            for(int j = 0; j < nnzB; j++) { //nnz should be number of non-zero element of B
              if (indicesB[j] == k && j >= ptrB[indicesA[i]] && j < ptrB[indicesA[i]+1]) {
                dot += dataA[i] * dataB[j];
              }
            }
          }
        result[row*num_rows+k] = dot;
        }
    }
}

__global__ void matvec_cudakernel_1perThread(REAL* matrix, REAL* vector, REAL *y, int num_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_rows) {
        y[i] = 0;
        REAL temp = 0.0;
        for (int j = 0; j < num_rows; j++)
            temp += matrix[i * num_rows + j] * vector[j];
        y[i] = temp;
    }
}

__global__ void matvec_cudakernel_1perThread_check_and_compute(REAL* matrix, REAL* vector, REAL *y, int num_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_rows) {
        y[i] = 0;
        REAL temp = 0.0;
        for (int j = 0; j < num_rows; j++) {
            if (matrix[i * num_rows + j] != 0.0) 
                temp += matrix[i * num_rows + j] * vector[j];
        }
        y[i] = temp;
    }
}


void spmv_cuda(const int num_rows, const int *ptrA, const int * indicesA, const REAL *dataA, const int *ptrB, const int * indicesB, const REAL *dataB,  REAL* result, int nnzA, int nnzB) {
  int *d_ptrA, * d_indicesA, *d_ptrB, * d_indicesB;
  REAL * d_dataA, * d_dataB, * d_result;

  hipMalloc(&d_ptrA, (num_rows+1)*sizeof(int));
  hipMalloc(&d_indicesA, nnzA*sizeof(int));

  hipMalloc(&d_ptrB, (num_rows+1)*sizeof(int));
  hipMalloc(&d_indicesB, nnzB*sizeof(int));


  hipMalloc(&d_dataA, nnzA*sizeof(REAL));
  hipMalloc(&d_dataB, nnzB*sizeof(REAL));

  hipMalloc(&d_result, num_rows * num_rows * sizeof(REAL));

  hipMemcpy(d_ptrA, ptrA, (num_rows+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_indicesA, indicesA, nnzA*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dataA, dataA, nnzA*sizeof(REAL), hipMemcpyHostToDevice);

  hipMemcpy(d_ptrB, ptrB, (num_rows+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_indicesB, indicesB, nnzB*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dataB, dataB, nnzB*sizeof(REAL), hipMemcpyHostToDevice);

  spmm_csr_kernel<<<256,256>>>(num_rows,d_ptrA, d_indicesA, d_dataA, d_ptrB, d_indicesB, d_dataB, d_result, nnzA, nnzB);
  hipMemcpy(result, d_result, num_rows*num_rows*sizeof(REAL), hipMemcpyDeviceToHost);
  //hipMemcpy(y_normal, d_y_normal, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_ptrA);
  hipFree(d_indicesA);
  hipFree(d_dataA);
  hipFree(d_ptrB);
  hipFree(d_indicesB);
  hipFree(d_dataB);
  hipFree(d_result);

}
