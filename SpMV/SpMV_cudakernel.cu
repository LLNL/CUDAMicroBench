#include "hip/hip_runtime.h"
#include "SpMV.h"

__global__ void spmv_csr_kernel(const int num_rows, const int *ptr, const int * indices, const REAL *data, const REAL * x, REAL *y)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < num_rows){
        REAL dot = 0;
        int row_start = ptr[row];
        int row_end = ptr[row+1];
       
        for(int n = row_start; n < row_end; n++){
           dot += data[n] * x[indices[n]];
        }
        y[row] += dot;
    }
}

__global__ void matvec_cudakernel_1perThread(REAL* matrix, REAL* vector, REAL *y, int num_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_rows) {
        y[i] = 0;
        REAL temp = 0.0;
        for (int j = 0; j < num_rows; j++)
            temp += matrix[i * num_rows + j] * vector[j];
        y[i] = temp;
    }
}

__global__ void matvec_cudakernel_1perThread_check_and_compute(REAL* matrix, REAL* vector, REAL *y, int num_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_rows) {
        y[i] = 0;
        REAL temp = 0.0;
        for (int j = 0; j < num_rows; j++) {
            if (matrix[i * num_rows + j] != 0.0) 
                temp += matrix[i * num_rows + j] * vector[j];
        }
        y[i] = temp;
    }
}


void spmv_cuda(const int num_rows, const int *ptr, const int * indices, const REAL *data, const REAL * x, REAL *y, int nnz, REAL* matrix, REAL *y_normal) {
  int *d_ptr, * d_indices;
  REAL * d_data, * d_x, *d_y, *d_matrix, *d_y_normal;

  hipMalloc(&d_ptr, (num_rows+1)*sizeof(int));
  hipMalloc(&d_indices, nnz*sizeof(int));

  hipMalloc(&d_data, nnz*sizeof(REAL));
  hipMalloc(&d_x, num_rows*sizeof(REAL));
  hipMalloc(&d_y, num_rows*sizeof(REAL));
  hipMalloc(&d_matrix, num_rows * num_rows * sizeof(REAL));

  hipMalloc(&d_y_normal, num_rows*sizeof(REAL));



  hipMemcpy(d_ptr, ptr, (num_rows+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_indices, indices, nnz*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_data, data, nnz*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, num_rows*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, matrix, num_rows*num_rows*sizeof(REAL), hipMemcpyHostToDevice);


  spmv_csr_kernel<<<256,256>>>(num_rows,d_ptr, d_indices, d_data, d_x, d_y);
  matvec_cudakernel_1perThread<<<256, 256>>>(d_matrix, d_x, d_y_normal, num_rows);
  matvec_cudakernel_1perThread_check_and_compute<<<256, 256>>>(d_matrix, d_x, d_y_normal, num_rows);
  hipMemcpy(y, d_y, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(y_normal, d_y_normal, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);

  hipFree(d_ptr);
  hipFree(d_indices);
  hipFree(d_data);
  hipFree(d_x);

  hipFree(d_y);
  hipFree(d_y_normal);
  hipFree(d_matrix);



}
