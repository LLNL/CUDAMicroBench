#include "hip/hip_runtime.h"
//******************************************************************************************************************//
// Copyright (c) 2021, University of North Carolina at Charlotte
// and Lawrence Livermore National Security, LLC.
// SPDX-License-Identifier: (BSD-3-Clause)
//*****************************************************************************************************************//
#include <vector>
#include <iostream>
#include <algorithm>
#include <ctime>

//sleep for the requested number of clocks
__global__ void timed_kernel( clock_t* clocksArray, int kernelIdx, int clockTicks ) {
    const clock_t start = clock();
    clock_t elapsed = 0;
    while( elapsed < clockTicks ) elapsed = clock() - start; 
    clocksArray[ kernelIdx ] = elapsed;
}

//parallel reduction: assume only one thread block used for computation;
//using more than a single block requires inter-block sychronization, see example 4.1/4.2 
__global__ void sum_clocks( clock_t* result, const clock_t* clocks, int numElements ) {
    const int CACHE_SIZE = 32; // equal to number of threads in thread block
    __shared__ clock_t cache[ CACHE_SIZE ];
    cache[ threadIdx.x ] = 0;
    if( threadIdx.x < numElements ) {
        for( int i = 0; i < numElements; i += CACHE_SIZE ) {
            cache[ threadIdx.x ] += clocks[ threadIdx.x + i ];   
        }
    }
    __syncthreads();
    for( int i = CACHE_SIZE / 2; i > 0; i /= 2 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
    }        
    result[ 0 ] = cache[ 0 ];
}

//------------------------------------------------------------------------------
int main( int , char**  ) {
    
    //first task: verify support for concurrent kernel execution
    hipDeviceProp_t prop = hipDeviceProp_t();
    int currentDevice = -1;
    hipGetDevice( &currentDevice );
    hipGetDeviceProperties( &prop, currentDevice );
    if( prop.concurrentKernels == 0 ) {
        std::cout << "Concurrent kernel execution not supported\n"
                  << "kernels will be serialized" << std::endl;
    }    

    // change this value to find the maximum number of concurrent kernels supported
    const int NUM_KERNELS = 8;
    const int NUM_CLOCKS  = NUM_KERNELS;
    const size_t CLOCKS_BYTE_SIZE = NUM_CLOCKS * sizeof( clock_t );
    const int KERNEL_EXECUTION_TIME_ms = 50; 
    float elapsed_time = 0.f;   
    hipEvent_t start, stop;
    std::vector< hipEvent_t >  kernel_events( NUM_KERNELS );
    hipStream_t time_compute_stream;
    std::vector< hipStream_t > kernel_streams( NUM_KERNELS );

    //create timing events
    hipEventCreate( &start );
    hipEventCreate( &stop  );

    //create kernel events
    for( std::vector< hipEvent_t >::iterator i =  kernel_events.begin();
         i != kernel_events.end(); ++i ) {
        hipEventCreateWithFlags( &(*i), hipEventDisableTiming );             
        
    }

    //create stream for time reporting kernel: stream must wait for all kernel events to be recorded 
    hipStreamCreate( &time_compute_stream );
    
    //create kernel streams
    for( std::vector< hipStream_t >::iterator i =  kernel_streams.begin();
        i != kernel_streams.end(); ++i ) {
        hipStreamCreate( &(*i) );           
    }

    //data array to hold timing information from kernel runs; TODO: use std::vector with page locked allocator
    clock_t* clocks    = 0;
    clock_t* clock_sum = 0; // sum of kernel execution times
    //we need host-allocated page locked memory because later-on an async memcpy operation is
    //is used; async operations *always* require page-locked memory
    hipHostAlloc( &clocks, CLOCKS_BYTE_SIZE, hipHostMallocPortable );
    hipHostAlloc( &clock_sum, sizeof( clock_t ), hipHostMallocPortable );
    clock_t* dev_clocks = 0;
    hipMalloc( &dev_clocks, CLOCKS_BYTE_SIZE );
    clock_t* dev_clock_sum = 0;
    hipMalloc( &dev_clock_sum, sizeof( clock_t ) );

    const int CLOCK_FREQ_kHz = prop.clockRate; 
    // BEGIN of async operations
    hipEventRecord( start, 0 );
    clock_t cpu_start = clock();
    for( int k = 0; k != NUM_KERNELS; ++k ) {
#ifdef FORCE_SERIALIZED
        // clock ticks = freq [s^-1] x time [s]  =
        //   10 ^ 3 x freq  x 10 ^ -3 time       =
        //   CLOCK_FREQ_kHz x KERNEL_EXECUTION_TIME_ms
        timed_kernel<<< 1, 1, 0, kernel_streams[ 0 ] >>>( dev_clocks,
                                                          k,
                                                          CLOCK_FREQ_kHz * KERNEL_EXECUTION_TIME_ms );
        if( k == NUM_KERNELS - 1 ) { // record event after all kernel have been executed
             hipEventRecord( kernel_events[ 0 ], kernel_streams[ 0 ] );
             hipStreamWaitEvent( time_compute_stream, kernel_events[ 0 ], 0 /*must be zero*/ );
        }
#else
        timed_kernel<<< 1, 1, 0, kernel_streams[ k ] >>>( dev_clocks,
                                                          k,
                                                          KERNEL_EXECUTION_TIME_ms * CLOCK_FREQ_kHz );
        hipEventRecord( kernel_events[ k ], kernel_streams[ k ] );
        hipStreamWaitEvent( time_compute_stream, kernel_events[ k ], 0 /*must be zero*/ );
#endif               
    }
    const int NUM_BLOCKS = 1;
    const int NUM_THREADS_PER_BLOCK = 32; // must match shared memory size
    const size_t SHARED_MEMORY_SIZE = 0;     
    
    sum_clocks<<< NUM_BLOCKS, NUM_THREADS_PER_BLOCK,
                  SHARED_MEMORY_SIZE, time_compute_stream >>>( dev_clock_sum, dev_clocks, NUM_KERNELS );
    hipMemcpyAsync( clock_sum, dev_clock_sum, sizeof( clock_t ), hipMemcpyDeviceToHost, time_compute_stream );
    hipMemcpyAsync( clocks, dev_clocks, CLOCKS_BYTE_SIZE, hipMemcpyDeviceToHost, time_compute_stream );
    
    //record event, not associated with any stream and therefore recorded
    //after *all* stream events are recorded
    hipEventRecord( stop, 0 );
    // END of async operations
    
    //sync everything
    //this synchronization call forces to wait until the stop event has been recorded;
    //the stop event is associated with the global context (the '0' in the cudaEventRegister call)
    //and therefore all events in the context must have been recorded before the stop event is recorded
    hipEventSynchronize( stop );
    const double cpu_elapsed_time = clock() - cpu_start;
    hipEventElapsedTime( &elapsed_time, start, stop );    
 
    //output information
    std::cout << "Clock:                                 " << double( CLOCK_FREQ_kHz ) * 1E-6 << " GHz" << std::endl; 
    std::cout << "Number of kernels:                     " << NUM_KERNELS << std::endl;
    std::cout << "Requested kernel execution time:       " << KERNEL_EXECUTION_TIME_ms << " ms" << std::endl;
    std::cout << "Computed kernel execution time:        " 
              << double( *std::max_element( clocks, clocks + NUM_KERNELS ) ) / CLOCK_FREQ_kHz << " ms" << std::endl;  
    std::cout << "Sum of kernel execution times:         " << double( *clock_sum ) / CLOCK_FREQ_kHz << " ms" << std::endl;  
    std::cout << "Total measured execution time:         " << elapsed_time << " ms" << std::endl;
    std::cout << "CPU elapsed time:                      " << 1000. * cpu_elapsed_time / CLOCKS_PER_SEC << " ms" << std::endl;
    //free resources
    for( std::vector< hipEvent_t >::iterator i =  kernel_events.begin();
         i != kernel_events.end(); ++i ) {
        hipEventDestroy( *i );            
    }

    //create sync stream: sync stream wait for all kernel events to be recorded 
    hipStreamDestroy( time_compute_stream );
    
    //create kernel streams
    for( std::vector< hipStream_t >::iterator i =  kernel_streams.begin();
         i != kernel_streams.end(); ++i ) {
        hipStreamDestroy( *i );           
    }

    hipHostFree( clock_sum );
    hipHostFree( clocks );
    hipFree( dev_clocks );
    hipFree( dev_clock_sum );

    //OPTIONAL, apparently it must be called in order for profiling and tracing tools
    //to show complete traces
    hipDeviceReset(); 

    return 0;

}
