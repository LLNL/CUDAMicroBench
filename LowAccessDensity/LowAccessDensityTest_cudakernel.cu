#include "hip/hip_runtime.h"
#include "LowAccessDensityTest.h"

__global__ 
void
LowAccessDensityTest_cudakernel(REAL* x, REAL* y, int n, REAL a, int stride)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (n/stride)) y[i] = a*x[i*stride];
}

__global__ 
void
LowAccessDensityTest_cudakernel_unified(REAL* x, REAL* y, int n, REAL a, int stride)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < (n/stride)) y[i] = a*x[i];
}

void LowAccessDensityTest_cuda(REAL* x, REAL* y, long int n, REAL a, int stride) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, (n/stride)*sizeof(REAL));

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, (n/stride)*sizeof(REAL), hipMemcpyHostToDevice);

  LowAccessDensityTest_cudakernel<<<(n+255)/256, 256>>>(d_x, d_y, n, a, stride);

  hipMemcpy(y, d_y, (n/stride)*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

void LowAccessDensityTest_cuda_unified(REAL* x, REAL* y, long int n, REAL a, int stride) {
  REAL *d_x, *d_y;
  hipMallocManaged(&d_x, (n/stride)*sizeof(REAL));
  hipMalloc(&d_y, (n/stride)*sizeof(REAL));

  for(int i = 0; i < (n/stride); i++) {
      d_x[i] = x[i*stride];
  }
  hipMemcpy(d_y, y, (n/stride)*sizeof(REAL), hipMemcpyHostToDevice);
  LowAccessDensityTest_cudakernel_unified<<<(n+255)/256, 256>>>(d_x, d_y, n, a, stride);
  hipMemcpy(y, d_y, (n/stride)*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);

}


