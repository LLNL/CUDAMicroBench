#include "hip/hip_runtime.h"
//******************************************************************************************************************//
// Copyright (c) 2021, University of North Carolina at Charlotte
// and Lawrence Livermore National Security, LLC.
// SPDX-License-Identifier: (BSD-3-Clause)
//*****************************************************************************************************************//
#include "axpy.h"

__global__ 
void
axpy_cudakernel_1perThread(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > 0 &&i < n) y[i] += a*x[i];
}

__global__ 
void
axpy_cudakernel_1perThread_misaligned(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    if (i < n) y[i] += a*x[i];
}

__global__ 
void
axpy_cudakernel_1perThread_warmup(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > 1 && i < n) y[i] += a*x[i];
}


void axpy_cuda(REAL* x, REAL* y, int n, REAL a) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice);
  
  //warm up
  axpy_cudakernel_1perThread_warmup<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();
  // Perform axpy elements
  axpy_cudakernel_1perThread_misaligned<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();
  axpy_cudakernel_1perThread<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  hipDeviceSynchronize();
  

  hipMemcpy(y, d_y, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

