#include "hip/hip_runtime.h"
#include "SpMM_csc.h"

__global__ void spmm_csc_kernel(const int num_rows, const int *ptrA, const int * indicesA, const REAL *dataA, const int *ptrB, const int * indicesB, const REAL *dataB,  REAL* result, int nnzA, int nnzB)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < num_rows){
        int row_start = ptrA[row];
        int row_end = ptrA[row+1];
        for(int column = 0; column < num_rows; column++){
            int column_start = ptrB[column];
            int column_end = ptrB[column+1];
            float dot = 0;
            for ( int i = row_start; i < row_end; i++) {
                for(int j = column_start; j < column_end; j++) {
                    if(indicesA[i] == indicesB[j]){
                          dot += dataA[i] * dataB[j];
                    }
                }
            }
         result[row*num_rows+column] = dot;
        }
    }
}

/*__global__ void matvec_cudakernel_1perThread(REAL* matrix, REAL* vector, REAL *y, int num_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_rows) {
        y[i] = 0;
        REAL temp = 0.0;
        for (int j = 0; j < num_rows; j++)
            temp += matrix[i * num_rows + j] * vector[j];
        y[i] = temp;
    }
}

__global__ void matvec_cudakernel_1perThread_check_and_compute(REAL* matrix, REAL* vector, REAL *y, int num_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_rows) {
        y[i] = 0;
        REAL temp = 0.0;
        for (int j = 0; j < num_rows; j++) {
            if (matrix[i * num_rows + j] != 0.0) 
                temp += matrix[i * num_rows + j] * vector[j];
        }
        y[i] = temp;
    }
}*/


void spmm_csr_cuda(const int num_rows, const int *ptrA, const int * indicesA, const REAL *dataA, const int *ptrB, const int * indicesB, const REAL *dataB,  REAL* result, int nnzA, int nnzB) {
  int *d_ptrA, * d_indicesA, *d_ptrB, * d_indicesB;
  REAL * d_dataA, * d_dataB, * d_result;

  hipMalloc(&d_ptrA, (num_rows+1)*sizeof(int));
  hipMalloc(&d_indicesA, nnzA*sizeof(int));

  hipMalloc(&d_ptrB, (num_rows+1)*sizeof(int));
  hipMalloc(&d_indicesB, nnzB*sizeof(int));


  hipMalloc(&d_dataA, nnzA*sizeof(REAL));
  hipMalloc(&d_dataB, nnzB*sizeof(REAL));

  hipMalloc(&d_result, num_rows * num_rows * sizeof(REAL));

  hipMemcpy(d_ptrA, ptrA, (num_rows+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_indicesA, indicesA, nnzA*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dataA, dataA, nnzA*sizeof(REAL), hipMemcpyHostToDevice);

  hipMemcpy(d_ptrB, ptrB, (num_rows+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_indicesB, indicesB, nnzB*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dataB, dataB, nnzB*sizeof(REAL), hipMemcpyHostToDevice);

  spmm_csc_kernel<<<256,256>>>(num_rows,d_ptrA, d_indicesA, d_dataA, d_ptrB, d_indicesB, d_dataB, d_result, nnzA, nnzB);
  //matvec_cudakernel_1perThread<<<256, 256>>>(d_matrix, d_x, d_y_normal, num_rows);
  //matvec_cudakernel_1perThread_check_and_compute<<<256, 256>>>(d_matrix, d_x, d_y_normal, num_rows);
  hipMemcpy(result, d_result, num_rows*num_rows*sizeof(REAL), hipMemcpyDeviceToHost);
  //hipMemcpy(y_normal, d_y_normal, num_rows*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_ptrA);
  hipFree(d_indicesA);
  hipFree(d_dataA);
  hipFree(d_ptrB);
  hipFree(d_indicesB);
  hipFree(d_dataB);
  hipFree(d_result);
}
